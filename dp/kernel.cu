#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <vector>
#include <string>
#include <iostream>
#include "fstream"
#include "iomanip" 
#include "traj.h"
#include "dp.h"
#include "proj_api.h"
#include "hip/hip_runtime.h"
#include ""
#include <assert.h>
#define TRAJ_NUM 10
#define POINTSUM 11731
//#define POINTSUM 120296
//#define POINTSUM 244982
//#define POINTSUM 359336
//#define POINTSUM 477684
//#define POINTSUM 605316
using namespace std;

void split(std::string& s, std::string& delim,std::vector< std::string >* ret)  
{  
	size_t last = 0;  
	size_t index=s.find_first_of(delim,last);  
	while (index!=std::string::npos)  
	{  
		ret->push_back(s.substr(last,index-last));  
		last=index+1;  
		index=s.find_first_of(delim,last);  
	}  
	if (index-last>0)  
	{  
		ret->push_back(s.substr(last,index-last));  
	}  
} 
__global__ void dp_kenel(float *x,float *y,bool *flag,int *count)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	int k = 0;
	for (int j = 0;j < i;j++)
		k+=count[j];
	DP(x,y,flag,k,k + count[i] - 1,50);
}

int main(void) {

	projPJ pj_merc, pj_latlong;

	if (!(pj_merc = pj_init_plus("+proj=merc +lon_0=0 +k=1 +x_0=0 +y_0=0 +ellps=WGS84 +datum=WGS84 +units=m +no_defs")))
		exit(1);
	if (!(pj_latlong = pj_init_plus("+proj=longlat +datum=WGS84 +no_defs")))
		exit(1);

	ifstream fin("C:\\Users\\Constantine\\Desktop\\out1.txt",ios::in);
	if(!fin)
	{
		cout<<"Cannot open input file!"<<endl;
		system("pause");
		return 1;
	}
	
	float x[POINTSUM] = {0.0};
	float y[POINTSUM] = {0.0};
	bool flag[POINTSUM] = {false};
	int traj_num[TRAJ_NUM];
	vector<std::string> vec_id;

	string line;
	string delim=" ";
	int trajIdx = 0;
	int point_Idx = 0;
	while(getline(fin,line)) {

		vector<string> result;
		split(line,delim,&result);

		vec_id.push_back(result.at(0).c_str());
		traj_num[trajIdx] = atoi(result.at(1).c_str());
		
		for ( vector<string>::iterator it = result.begin()+2;it != result.end(); it++) {

			double lng = atof(&(*it->c_str()));
			it++;
			double lat = atof(&(*it->c_str()));

			lat *= DEG_TO_RAD;
			lng *= DEG_TO_RAD;
			pj_transform(pj_latlong, pj_merc, 1, 1, &lng, &lat, NULL);

			
			x[point_Idx] = lng;
			y[point_Idx] = lat;
			point_Idx++;
		}
		trajIdx++;
		if (trajIdx == 10)
		    break;
	}
	fin.close();

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	float *d_x ;
	float *d_y ;
    bool *d_flag;
	int *d_count;

	cudaStatus = hipMalloc((void**)&d_x, POINTSUM * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_y, POINTSUM * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_flag, POINTSUM * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_count, TRAJ_NUM * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_x, x, POINTSUM * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(d_y, y, POINTSUM * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(d_flag, flag, POINTSUM * sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(d_count, traj_num, TRAJ_NUM * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	clock_t starttime = clock();
	dp_kenel<<<2,5>>>(d_x,d_y,d_flag,d_count);   //���к˺���
	hipDeviceSynchronize();
	clock_t endtime = clock();

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	float o_x[POINTSUM] ={0.0};
	float o_y[POINTSUM] ={0.0};
	bool o_flag[POINTSUM] = {false};

	cudaStatus = hipMemcpy(o_x, d_x, POINTSUM * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(o_y, d_y, POINTSUM * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(o_flag, d_flag, POINTSUM * sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}
Error:
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_flag);
	hipFree(d_count);
	hipFree(o_x);
	hipFree(o_y);
	hipFree(o_flag);

	/*int simple_Num = 0;
	for (int i=0;i< POINTSUM;i++)
	{
	if (o_flag[i]==true)
	{
	cout<<o_y[i]<<endl;
	simple_Num++;
	}
	}
	cout << simple_Num << endl;*/

	ofstream fout("C:\\Users\\Constantine\\Desktop\\10.txt",ios::out);
	int j = 0;
	int i ;
	int k = 0;
	for (i = 0;i<10;i++)
	{
		for(;j<traj_num[i]+k;j++)
		{
			if(o_flag[j] == true)
			{
				double x = (double)o_x[j];
				double y = (double)o_y[j];
				pj_transform(pj_merc, pj_latlong, 1, 1, &x, &y, NULL);
				x /= DEG_TO_RAD;
				y /= DEG_TO_RAD;
				fout<<fixed<<setprecision(6)<<vec_id.at(i)<<","<<x<<","<<y<<endl;
			}
		}
		k = j;
	}
	fout.close();
		
	printf("totals time is %lf s\n",(double)(endtime-starttime)/CLOCKS_PER_SEC);
	system("pause");
}